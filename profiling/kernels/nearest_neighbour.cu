
#include <hip/hip_runtime.h>
__device__ __forceinline__ float square_dist(float3 a, float3 b) {
    float dx = (a.x - b.x);
    float dy = (a.y - b.y);
    float dz = (a.z - b.z);
    return dx * dx + dy * dy + dz * dz;
}

__global__ void find_min_distance_index(const float *points, const float *query, int *min_idx,
                                        float *min_distance, int num_points) {
    __shared__ float shared_distances[1024];
    __shared__ int shared_idx[1024];

    int global_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;
    if (global_idx < num_points) {
        shared_distances[threadIdx.x] = square_dist(
            make_float3(
                points[global_idx * 3],
                points[global_idx * 3 + 1],
                points[global_idx * 3 + 2]
            ), make_float3(
                query[0],
                query[1],
                query[2]
            ));
        shared_idx[threadIdx.x] = global_idx;
    }
    __syncthreads();

    // Perform reduction to find the minimum
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0) {
            if (shared_distances[tid + stride] < shared_distances[tid]) {
                shared_distances[tid] = shared_distances[tid + stride];
                shared_idx[tid] = shared_idx[tid + stride];
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        min_distance[blockIdx.x] = shared_distances[0];
        min_idx[blockIdx.x] = shared_idx[0];
    }
}
