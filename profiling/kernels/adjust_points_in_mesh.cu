
#include <hip/hip_runtime.h>
__device__ __inline__ float3 cross_product(float3 a, float3 b) {
    return make_float3(
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    );
}

__device__ __inline__ float dot_product(float3 a, float3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ __inline__ float3 subtract(float3 a, float3 b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ __inline__ float3 add(float3 a, float3 b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}


__device__ __inline__ float square_dist(float3 a, float3 b) {
    return (a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y) + (a.z - b.z) * (a.z - b.z);
}

__device__ __inline__ float clamp(float value) {
    return max(0.0f, min(value, 1.0f));
}

__device__ __inline__ float3 scalar_multiply(float3 p, float scalar) {
    return make_float3(p.x * scalar, p.y * scalar, p.z * scalar);
}

__device__ __inline__ float3 get_v0(float *triangles, int idx) {
    return make_float3(
        triangles[idx * 9],
        triangles[idx * 9 + 1],
        triangles[idx * 9 + 2]
    );
}

__device__ __inline__ float3 get_edge1(float *triangles, int idx) {
    return make_float3(
        triangles[idx * 9 + 3],
        triangles[idx * 9 + 4],
        triangles[idx * 9 + 5]
    );
}

__device__ __inline__ float3 get_edge2(float *triangles, int idx) {
    return make_float3(
        triangles[idx * 9 + 6],
        triangles[idx * 9 + 7],
        triangles[idx * 9 + 8]
    );
}


__device__ int ray_intersects_triangle(float3& query, const float3& dir,
                                       float3 v0, float3 edge1, float3 edge2) {
    const float EPSILON = 1e-7f;
    float3 b;
    float v;

    float3 a = cross_product(dir, edge2);
    float det = dot_product(edge1, a);
    if (fabsf(det) < EPSILON)
        return 0;

    float3 c = subtract(query, v0);
    float u = dot_product(a, c);
    if (det > EPSILON) {
        if (u < 0 || u > det)
            return 0;

        b = cross_product(c, edge1);
        v = dot_product(b, dir);
        if (v < 0 || u + v > det)
            return 0;
    } else {
        if (u > 0 || u < det)
            return 0;

        b = cross_product(c, edge1);
        v = dot_product(b, dir);
        if (v > 0 || u + v < det)
            return 0;
    }

    float inv_det = 1.0f / det;
    float t = inv_det * dot_product(edge2, b);

    if (t < EPSILON || t > 1 - EPSILON)
        return 0;

    return 1;
}

__device__ float3 closest_point_on_triangle(float3& query,
                                            float3 v0, float3 edge1, float3 edge2) {
    const float EPSILON = 1e-10f;
    float3 q_to_v0 = subtract(query, v0);

    float d1 = dot_product(edge1, q_to_v0);
    float d2 = dot_product(edge2, q_to_v0);
    float d3 = dot_product(edge1, edge1);
    float d4 = dot_product(edge1, edge2);
    float d5 = dot_product(edge2, edge2);

    float denom = d3 * d5 - d4 * d4 + EPSILON;
    float v_clamped = clamp((d5 * d1 - d4 * d2) / denom);
    float w_clamped = clamp((d3 * d2 - d4 * d1) / denom);
    float u_clamped = clamp(1 - v_clamped - w_clamped);

    float sum_clamped = u_clamped + v_clamped + w_clamped;
    float3 u_vec = scalar_multiply(v0, u_clamped / sum_clamped);
    float3 v_vec = scalar_multiply(add(v0, edge1), v_clamped / sum_clamped);
    float3 w_vec = scalar_multiply(add(v0, edge2), w_clamped / sum_clamped);

    return add(w_vec, add(u_vec, v_vec));
}

__global__ void adjust_point_in_mesh(float *triangles, int num_triangles,
                                     float *points, int num_points,
                                     float *normals, float *centers) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= num_points) return;

    float3 query = make_float3(
        points[pt_idx * 3],
        points[pt_idx * 3 + 1],
        points[pt_idx * 3 + 2]
    );
    const float3 ray_dir = make_float3(1.0f, 1.0f, 1.0f);

    int hit_count = 0;
    for (int tri_idx = 0; tri_idx < num_triangles; ++tri_idx) {
        hit_count += ray_intersects_triangle(
            query, ray_dir, get_v0(triangles, tri_idx),
            get_edge1(triangles, tri_idx), get_edge2(triangles, tri_idx)
        );
    }

    if (hit_count % 2 == 1) {
        float closest_distance_sq = 1e20;
        int closest_index = 0;

        for (int tri_idx = 0; tri_idx < num_triangles; ++tri_idx) {
            // Eliminate possibilities with triangle inequality
            float dist_to_center = sqrtf(square_dist(
                make_float3(
                    centers[tri_idx * 4],
                    centers[tri_idx * 4 + 1],
                    centers[tri_idx * 4 + 2]
                ), query));
            float radius = centers[tri_idx * 4 + 3];
            if (closest_distance_sq < dist_to_center - radius) {
                continue;
            }

            // Find closest point on triangle
            float3 closest_point = closest_point_on_triangle(
                query, get_v0(triangles, tri_idx),
                get_edge1(triangles, tri_idx), get_edge2(triangles, tri_idx)
            );
            float distance = square_dist(closest_point, query);

            if (distance < closest_distance_sq) {
                closest_index = tri_idx;
                closest_distance_sq = distance;
            }
        }

        float3 normal = make_float3(
            normals[closest_index * 3],
            normals[closest_index * 3 + 1],
            normals[closest_index * 3 + 2]
        );
        float3 adjustment = scalar_multiply(normal, sqrtf(closest_distance_sq));

        points[pt_idx * 3] += adjustment.x;
        points[pt_idx * 3 + 1] += adjustment.y;
        points[pt_idx * 3 + 2] += adjustment.z;
    }
}
